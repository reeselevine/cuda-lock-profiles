
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <string>
#include <vector>

const int minWorkgroups = 2;
const int maxWorkgroups = 2;
const int numIterations = 10;
const int expectedCount = 20480;

// general
int* var;
int* d_var;

// spin lock
int* flag;
int* d_flag;

// petersons
int* level;
int* d_level;
int* victim;
int* d_victim;

__global__
void petersons(volatile int* _level, volatile int* _victim, int* _var, int numIterations) {
	if (threadIdx.x == 0) {
		for (int i = 0; i < numIterations; i++) {
			for (int j = 0; j < gridDim.x - 1; j++) {
				_level[blockIdx.x] = j;
				_victim[j] = blockIdx.x;
				for (int k = 0; k < gridDim.x; k++) {
					while (k != blockIdx.x && _level[k] >= j && _victim[j] == blockIdx.x);
				}
			}
			__threadfence();
			*_var = *_var + 1;
			__threadfence();
			_level[blockIdx.x] = -1;
		}
	}
}

__global__
void spinLock(volatile int* _flag, int* _var, int numIterations) {
	if (threadIdx.x == 0) {
		for (int i = 0; i < numIterations; i++) {
			while(atomicCAS((int*) _flag, 0, 1) == 1);
			__threadfence();
			*_var = *_var + 1;
			__threadfence();
			*_flag = 0;
		}
	}
}

void initializeBuffers(std::string testName) {
	var = (int*)malloc(1*sizeof(int));
	hipMalloc(&d_var, 1*sizeof(int));
	if (testName == "spin-lock") {
		flag = (int*)malloc(1*sizeof(int));
		hipMalloc(&d_flag, 1*sizeof(int));
	} else if (testName == "petersons") {
		level = (int*)malloc(maxWorkgroups*sizeof(int));
		hipMalloc(&d_level, maxWorkgroups*sizeof(int));
		victim = (int*)malloc(maxWorkgroups*sizeof(int));
		hipMalloc(&d_victim, maxWorkgroups*sizeof(int));
	}
}

void prepareBuffers(std::string testName) {
	if (testName == "spin-lock") {
		*flag = 0;
		hipMemcpy(d_flag, flag, 1*sizeof(int), hipMemcpyHostToDevice);
	} else if (testName == "petersons") {
		for (int i = 0; i < maxWorkgroups; i++) {
			level[i] = 0;
			victim[i] = 0;
		}
		hipMemcpy(d_level, level, maxWorkgroups*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_victim, victim, maxWorkgroups*sizeof(int), hipMemcpyHostToDevice);
	}
}

void freeBuffers(std::string testName) {
	hipFree(d_var);
	free(var);
	if (testName == "spin-lock") {
		hipFree(d_flag);
		free(flag);
	} else if (testName == "petersons") {
		hipFree(d_level);
		hipFree(d_victim);
		free(level);
		free(victim);
	}
}

void runTest(std::string testName, int iterationsPerTest, int numWorkgroups) {
	if (testName == "spin-lock") {
		std::cout << "iterations per test: " << iterationsPerTest << "\n";
		spinLock<<<numWorkgroups, 1>>>(d_flag, d_var, iterationsPerTest);
	} else if (testName == "petersons") {
		petersons<<<numWorkgroups, 1>>>(d_level, d_victim, d_var, iterationsPerTest);
	}
}


int main(int argc, char* argv[]) {
	if (argc != 2) {
		std::cout << "Test name must be specified\n";
	}
	std::string testName(argv[1]);
	srand (time(NULL));

	std::cout << "Running Test" << testName << "\n";
	initializeBuffers(testName);
	double sum = 0;
	std::chrono::time_point<std::chrono::system_clock> start, end;
	for (int numWorkgroups = minWorkgroups; numWorkgroups <= maxWorkgroups; numWorkgroups*=2) {
		std::cout << "\nTest workgroups " << numWorkgroups << "\n";
		int iterationsPerTest = expectedCount/numWorkgroups;
		for (int i = 0; i < numIterations + 1; i++) {
			std::cout << "\ntest iteration " << i << "\n";
			*var = 0;
			hipMemcpy(d_var, var, 1*sizeof(int), hipMemcpyHostToDevice);
			prepareBuffers(testName);
			start = std::chrono::system_clock::now();
		        runTest(testName, iterationsPerTest, numWorkgroups);
			end = std::chrono::system_clock::now();
			hipMemcpy(var, d_var, 1*sizeof(int), hipMemcpyDeviceToHost);
			std::chrono::duration<double> result = end - start;
			if (i > 0) sum += result.count();
			std::cout << "iteration time: " << result.count() << "s\n";
			std::cout << "expected: " << expectedCount << ", actual: " << *var << "\n";
			if (expectedCount != *var) {
				std::cout << "Expected not equal to actual!\n";
			}
		}
		std::cout << "Average test iteration time: " << sum / numIterations << "s\n";
		sum = 0;
	}
	freeBuffers(testName);
	return 0;
}
